#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

#define TRUE 1

unsigned int EMPTY = UINT_MAX;

char str[200];
int i, j, k, e, p, na;
int nb; // numero de blocos a serem usados no kernel
FILE *fp;
int line=1;
int nnos, idx_ni, nfol; // numero de nos, indice de nos internos, numero de folhas
int hnnos; // tamanho da tabela hash
int ennos; // tamanho do vetor com as distancias entre as especies (matriz triangular superior)
int pos_ins, n_ins; // posicao de insercoes e numero de insercoes
int *nz; // contem indice do no; para os nos a serem inseridos, contem o indice do ponto de insercao
		 // para os nos internos a serem usados na insercao, contem -2
float *nz_br; // distancia do ramo (branch)
float *nz_dr; // distancias ate o no raiz
float *nz_de; // distancias entre especies
int *nz_qf; // altura do no
int *nz_qe; // quantidade de especies abaixo do no
int *nz_p; // pai dono
int *nz_f1; // filho da esquerda do no
int *nz_f2; // filho da direita do no
unsigned int *nz_sig; // assinatura do no - da o caminho em bits ate o raiz
unsigned int *nz_hsig; // hash da assinatura do no
unsigned int *nz_hval; // indice do no na tabela hash
long long GPU_start_time;
long long GPU_time;

// pointers to GPU memory
int *nz_d;
float *nz_br_d;
float *nz_dr_d;
float *nz_de_d;
int *nz_qf_d;
int *nz_qe_d;
int *nz_p_d;
int *nz_f1_d;
int *nz_f2_d;
unsigned int *nz_sig_d;
unsigned int *nz_hsig_d;
unsigned int *nz_hval_d;
//int pos_ins_d, idx_ni_d;
//
char *symb, **nz_sy, **n_arq;
char str_tmp[100];
char str_float[30];
int nbint, nbuint, nbhuint, nbfloat, nbefloat; // tamanho em bytes dos tipos basicos
hiprandState *seed_d;
float zero = 0.0; // para facilitar impressao da matriz de distancias
char arquivo[100];

// Forward function declarations
long long start_timer();
long long stop_timer(long long start_time, char *name);

// print tree in newick format
char *toNewick(int raiz);

// find next prime number greater than n
int nextprime( int n );

// kernel
__global__ void Mutate_tree_Gpu(int nnos, int hnnos, int pos_ins, int idx_ni, int *nz, float *nz_br, float *nz_dr, float *nz_de, int *nz_qf, int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, unsigned int *nz_sig, unsigned int *nz_hsig, unsigned int *nz_hval, hiprandState *states, unsigned long seed);

// auxiliary kernel functions
__device__ int quadratic_probing_insert(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int val, int hnnos);
__device__ int quadratic_probing_search(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int hnnos);

// Main program
int main()
{

	symb = (char *) malloc(50);

	fp = fopen("wellParser.out", "r");
	if (fp == NULL) {
		printf("\nCannot open file\n");
		exit(0);
	}
	
	fscanf(fp,"%d %d", &nb, &nnos); // numero de (arvores) blocos a serem usados no kernel
	printf("Num. arvores: %d, cada uma com %d nós\n", nb, nnos);

	nfol = nnos / 2;
	nz = (int *) malloc(nnos * sizeof(int));
	nz_sy = (char **) malloc(nnos * sizeof(char *));
	nz_dr = (float *) malloc(nnos * sizeof(float));	
	ennos = (nfol * (nfol - 1)) / 2;
	nz_de = (float *) malloc(ennos * sizeof(float));
	nz_br = (float *) malloc(nnos * sizeof(float));
	nz_qf = (int *) malloc(nnos * sizeof(int));
	nz_qe = (int *) malloc(nnos * sizeof(int));
	nz_p = (int *) malloc(nnos * sizeof(int));
	nz_f1 = (int *) malloc(nnos * sizeof(int));
	nz_f2 = (int *) malloc(nnos * sizeof(int));	
	hnnos = nextprime(2*nnos);
	nz_sig = (unsigned int *) malloc(hnnos * sizeof(unsigned int));
	nz_hsig = (unsigned int *) malloc(hnnos * sizeof(unsigned int));
	nz_hval = (unsigned int *) malloc(hnnos * sizeof(unsigned int));
	n_arq = (char **) malloc(nb * sizeof(char *));	  // guarda nome dos arquivos das arvores

	nbint = nnos * sizeof(int);
	nbuint = nnos * sizeof(unsigned int);
	nbhuint = hnnos * sizeof(unsigned int);
	nbfloat = nnos * sizeof(float);
	nbefloat = ennos * sizeof(float);
	
	hipMalloc((void **)&nz_d, nb * nbint);
    hipMalloc((void **)&nz_br_d, nb * nbfloat);
    hipMalloc((void **)&nz_dr_d, nb * nbfloat);
    hipMalloc((void **)&nz_de_d, nb * nbefloat);
    hipMalloc((void **)&nz_qf_d, nb * nbint);
    hipMalloc((void **)&nz_qe_d, nb * nbint);
    hipMalloc((void **)&nz_p_d, nb * nbint);
    hipMalloc((void **)&nz_f1_d, nb * nbint);
    hipMalloc((void **)&nz_f2_d, nb * nbint);
    hipMalloc((void **)&nz_sig_d, nb * nbuint);
    hipMalloc((void **)&nz_hsig_d, nb * nbhuint);
    hipMalloc((void **)&nz_hval_d, nb * nbhuint);
    hipMalloc((void **)&seed_d, nb * nnos*sizeof(hiprandState));
    
    if( nz_d==0 || nz_br_d==0 || nz_dr_d==0 || nz_de_d==0 || nz_qf_d==0 || nz_qe_d==0 || nz_p_d==0 || nz_f1_d==0 || nz_f2_d==0 || nz_sig_d==0 || nz_hsig_d==0 || nz_hval_d==0 ) {
      printf("couldn't allocate memory\n"); 
      return 1;
	} 

////
	
	for(na=0; na < nb; na++) { 			// na = numero de arvores
	
		fscanf(fp,"%s", symb);
		if (na == 0) { printf("Arquivo: %s\n", symb); }
		n_arq[na] = (char *) malloc(50);
		strcpy(n_arq[na], symb);
	
		fscanf(fp,"%d %d", &nnos, &idx_ni);
		if (na == 0) { printf("No nos: %d, Indice no interno: %d\n", nnos, idx_ni); }
		fscanf(fp,"%d %d", &pos_ins, &n_ins);
		if (na == 0) { printf("Pos inic: %d, No insercoes: %d\n", pos_ins, n_ins); }

		if (na == 0) { printf("Arvore: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz[i]);
			if (na == 0) { printf("%d ", nz[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("Simbolos: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%s", symb);
			nz_sy[i] = (char *) malloc(50);
			strcpy(nz_sy[i], symb);
			if (na == 0) { printf("%s ", nz_sy[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("Ramos: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%f", &nz_br[i]);
			if (na == 0) { printf("%f ", nz_br[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("No Filhos: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz_qf[i]);
			if (na == 0) { printf("%d ", nz_qf[i]); }
		}
		if (na == 0) { printf("\n"); }
	
		if (na == 0) { printf("No Especies: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz_qe[i]);
			if (na == 0) { printf("%d ", nz_qe[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("Pais: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz_p[i]);
			if (na == 0) { printf("%d ", nz_p[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("Filhos 1: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz_f1[i]);
			if (na == 0) { printf("%d ", nz_f1[i]); }
		}
		if (na == 0) { printf("\n"); }

		if (na == 0) { printf("Filhos 2: "); }
		for(i=0; i<nnos; i++) {
			fscanf(fp,"%d", &nz_f2[i]);
			if (na == 0) { printf("%d ", nz_f2[i]); }
		}
		if (na == 0) { printf("\n"); }
	
		for(i=0; i<hnnos; i++) {
			nz_sig[i] = 0;
		}
	
		for(i=0; i<hnnos; i++) {
			nz_hsig[i] = (unsigned int) EMPTY;
			nz_hval[i] = (unsigned int) EMPTY;
		}

		if (na == 0) { toNewick(nnos-1); printf(";\n"); }

		if (na > 0) {
			for(j = 0; j < nnos; j++) {
				if (nz[j] >= 0) nz[j] = nz[j] + na*nnos;
				if (nz_p[j] >= 0) nz_p[j] = nz_p[j] + na*nnos;
				if (nz_f1[j] >= 0) nz_f1[j] = nz_f1[j] + na*nnos;
				if (nz_f2[j] >= 0) nz_f2[j] = nz_f2[j] + na*nnos;
			}
			nz[nfol] = -na;
		}
			// move data to GPU
		hipMemcpy(nz_d + (na*nnos), nz, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_br_d + (na*nnos), nz_br, nbfloat, hipMemcpyHostToDevice);
		hipMemcpy(nz_dr_d + (na*nnos), nz_dr, nbfloat, hipMemcpyHostToDevice);
		hipMemcpy(nz_de_d + (na*ennos), nz_de, nbefloat, hipMemcpyHostToDevice);
		hipMemcpy(nz_qf_d + (na*nnos), nz_qf, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_qe_d + (na*nnos), nz_qe, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_p_d + (na*nnos), nz_p, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_f1_d + (na*nnos), nz_f1, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_f2_d + (na*nnos), nz_f2, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_sig_d + (na*nnos), nz_sig, nbuint, hipMemcpyHostToDevice);
		hipMemcpy(nz_hsig_d + (na*hnnos), nz_hsig, nbhuint, hipMemcpyHostToDevice);
		hipMemcpy(nz_hval_d + (na*hnnos), nz_hval, nbhuint, hipMemcpyHostToDevice);
	}

	fclose(fp);

//	
	GPU_start_time = start_timer();
	
	// call kernel
	Mutate_tree_Gpu<<<nb, nfol>>>(nnos, hnnos, pos_ins, idx_ni, nz_d, nz_br_d, nz_dr_d, nz_de_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d, nz_sig_d, nz_hsig_d, nz_hval_d, seed_d, time(NULL));

	hipDeviceSynchronize(); // this is only needed for timing purposes	
	GPU_time = stop_timer(GPU_start_time, "\t Total");
//	
	
	p = nb-(nb/2)+(nb/4)-1;

	// copy data back to the CPU
	hipMemcpy(nz, nz_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_br, nz_br_d+p*nnos, nbfloat, hipMemcpyDeviceToHost);
	hipMemcpy(nz_dr, nz_dr_d+p*nnos, nbfloat, hipMemcpyDeviceToHost);
	hipMemcpy(nz_de, nz_de_d+p*ennos, nbefloat, hipMemcpyDeviceToHost);
	hipMemcpy(nz_qf, nz_qf_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_qe, nz_qe_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_p, nz_p_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_f1, nz_f1_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_f2, nz_f2_d+p*nnos, nbint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_sig, nz_sig_d+p*nnos, nbuint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_hsig, nz_hsig_d+p*hnnos, nbhuint, hipMemcpyDeviceToHost);
	hipMemcpy(nz_hval, nz_hval_d+p*hnnos, nbhuint, hipMemcpyDeviceToHost);
	
	if (p > 0) {
		for(j = 0; j < nnos; j++) {
			if (nz[j] >= 0) nz[j] = nz[j] - p*nnos;
			if (nz_p[j] >= 0) nz_p[j] = nz_p[j] - p*nnos;
			if (nz_f1[j] >= 0) nz_f1[j] = nz_f1[j] - p*nnos;
			if (nz_f2[j] >= 0) nz_f2[j] = nz_f2[j] - p*nnos;
		}
	}

	printf("Arquivo: %s\n", n_arq[p]);

	toNewick(nnos-1);
	printf(";\n");
//	
	printf("Arvore: ");
	for(i=0; i<nnos; i++) {
		printf("%d ", nz[i]);
	}
	printf("\n");
//
	printf("Pais: ");
	for(i=0; i<nnos; i++) {
		printf("%d ", nz_p[i]);
	}
	printf("\n");

	printf("f1: ");
	for(i=0; i<nnos; i++) {
		printf("%d ", nz_f1[i]);
	}
	printf("\n");

	printf("f2: ");
	for(i=0; i<nnos; i++) {
		printf("%d ", nz_f2[i]);
	}
	printf("\n");

	printf("Dst Raiz: ");
	for(i=0; i<nnos; i++) {
		if (i == nfol) continue; // desconta o no da posicao nfol
		printf("%.2f ", nz_dr[i]); // pois este nao e usado
	}
	printf("\n");

	printf("Assinatura: ");
	for(i=0; i<nnos; i++) {
		if (i == nfol) continue;
		printf("%u ", nz_sig[i]);
	}
	printf("\n");
	
	printf("Hash Sign: ");
	for(i=0; i<hnnos; i++) {
		if (i == nfol) continue;
		printf("%u ", nz_hsig[i]);
	}
	printf("\n");
	
	printf("Hash Val: ");
	for(i=0; i<hnnos; i++) {
		if (i == nfol) continue;
		printf("%u ", nz_hval[i]);
	}
	printf("\n");

	e = 0; // indexa a matriz triangular superior (representada num array) que contem a distancia
		   // entre as especies
	printf("Distancias: \n");
	printf("%7s ", nz_sy[0]);
	for(i=1; i<nfol; i++)
		printf("%4s\t", nz_sy[i]);
	printf("\n");
	for(i=0; i<nfol; i++) {
		for(j=0; j<=i; j++)
			printf("%.5f\t", zero);
		for(k=i+1; k<nfol; k++) {
			printf("%.5f\t", nz_de[e]);
			e++;
		}
		printf("\n");
	}
	printf("\n");
//		
	free(nz);
	free(nz_br);
	free(nz_dr);
	free(nz_de);
	free(nz_p);
	free(nz_f1);
	free(nz_f2);
	free(nz_sig);
	free(nz_hsig);
	free(nz_hval);
	free(symb);
	free(nz_sy);
	free(nz_qf);
	free(nz_qe);
//	
	hipFree(nz_d);    
    hipFree(nz_br_d);
    hipFree(nz_dr_d);
    hipFree(nz_de_d);
    hipFree(nz_qf_d);
    hipFree(nz_qe_d);
    hipFree(nz_p_d);    
    hipFree(nz_f1_d);
    hipFree(nz_f2_d);
    hipFree(nz_sig_d);
    hipFree(nz_hsig_d);
    hipFree(nz_hval_d);
//    
    return 0;
}

// Returns the current time in microseconds
long long start_timer() {
struct timeval tv;
gettimeofday(&tv, NULL);
return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *name) {
struct timeval tv;
gettimeofday(&tv, NULL);
long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
return end_time - start_time;
}

char *toNewick(int idRaiz) {
  
    strcpy(str_tmp,"");
    strcpy(str_float,"");
		
	if (nz_f1[idRaiz] < 0) { // Não tem filhos 
		strcat (str_tmp, nz_sy[idRaiz]);
		strcat (str_tmp, ":");
//		sprintf(str_float,"%0.2f", nz_br[idRaiz]); 
		sprintf(str_float,"%f", nz_br[idRaiz]); 
		strcat (str_tmp, str_float);
		return str_tmp;
	} else { // Tem filhos 
		printf("(");
		printf("%s", toNewick(nz_f1[idRaiz]));
		printf(",");
		printf("%s", toNewick(nz_f2[idRaiz]));
		printf(")");
		printf("%s", nz_sy[idRaiz]);
		printf(":");
//		sprintf(str_float,"%0.2f", nz_br[idRaiz]);
		sprintf(str_float,"%f", nz_br[idRaiz]); 
		printf("%s", str_float);
		return "";
	}
}

int nextprime( int n ) {
	int Divisor, PossiblePrime;
	int FoundPrime;

	PossiblePrime = n;
	if( PossiblePrime <= 2 )
		PossiblePrime = 2;
	else
		if( PossiblePrime != 3 ) {
			if( PossiblePrime % 2 == 0 )
				PossiblePrime++;	/* Need An Odd Number */
			for( ; ; PossiblePrime += 2 ) {
				FoundPrime = !TRUE;
				for( Divisor = 3; PossiblePrime % Divisor; Divisor += 2 )
					if( Divisor * Divisor > PossiblePrime ) {
						FoundPrime = TRUE;
						break;
					}
				if( FoundPrime )
					break;
			}
		}
	return PossiblePrime;
}

__device__ int quadratic_probing_insert(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int val, int hnnos) {
    unsigned int j, hk, old;
    
    int ib = blockIdx.x; // identificador do bloco
    j = 0;
    hk = sig  % hnnos;
    while(j < hnnos) {
    	old = atomicCAS(&nz_hsig[hk+ib*hnnos], UINT_MAX, sig); // se posicao estiver vazia (UINT_MAX = EMPTY)
		if (old == UINT_MAX) {
			nz_hval[hk+ib*hnnos] = val;
			return (hk+ib*hnnos);
    	}
        j++;
        hk = (hk + j * j) % hnnos;
//        hk = (hk + j) % hnnos;
    }
    return (-1);
}

__device__ int quadratic_probing_search(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int hnnos) {
    unsigned int j, hk;
    
    int ib = blockIdx.x; // identificador do bloco
    j = 0;
    hk = sig  % hnnos;
    while(j < hnnos) {
		if (nz_hsig[hk+ib*hnnos] == sig) {
			return (nz_hval[hk+ib*hnnos]);
    	}
        j++;
        hk = (hk + j * j) % hnnos;
//        hk = (hk + j) % hnnos;
    }
    return (-1);
}

// estas duas funcoes sao usada para mapear os indices de um array para uma matriz triangular 
// superior correspondente (sem a diagonal). para uma matriz nxn, o array terá n(n-1)/2 elementos

__host__ __device__ int row_index( int i, int M ){ // retorna o indice da linha
	M--;
    float m = M;
    float row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(float)i - 7) )) / -2;
    if( row == (float)(int) row ) row -= 1;
    return (int) row;
}

__host__ __device__ int column_index( int i, int M ){ // retorna o indice da coluna
    int row = row_index( i, M);
    M--;
    return 1 + (i - M * row + row*(row+1) / 2);
}


__global__ void Mutate_tree_Gpu(int nnos, int hnnos, int pos_ins, int idx_ni, int *nz, float *nz_br, float *nz_dr, float *nz_de, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, unsigned int *nz_sig, unsigned int *nz_hsig, unsigned int *nz_hval, hiprandState *states, unsigned long seed) {
	float y; // acumula soma das arestas
	int nfol; // numero de folhas da arvore
	int j; // indice para thread ativa
	int a, b; // usados no calculo da faixa de elementos (da matriz triangular) a serem considerados
	unsigned int sig1, sig2, sig3, sig4; // assinaturas de tres nos - da o caminho em bits ate o raiz
	int bit; // contem bit sendo analizado
	int ancc; // indice do ancestral comum
	int nthreads; // numero de threads ativas
	int r, c; // linha e coluna da matriz triangular superior
	int bits; // conta quantos bits sao iguais
	int i = threadIdx.x; // identificador da thread
	int ib = blockIdx.x; // identificador do bloco
	int it; // indice de acesso global das threads
	int ennos; // tamanho da matriz de distancias
	
	nfol = nnos / 2; // folhas estao na metade inferior
	ennos = (nfol * (nfol - 1)) / 2;
	it = i + ib*nnos;
	
	if (i < nfol) { // nos folhos calculam distancia ate a raiz e armazena o caminho (assinatura
					// em bits) até a raiz
		y = 0;
		j = it;  // associa threads com nos folhas
		nz_sig[it] = 1;
		while (j != -1) {
			y = y + nz_br[j]; // acumula a distancia
			if (nz_p[j] == -1) 
				break;
			nz_sig[it] <<= 1;  // acumula o caminho
			if (nz_f1[nz_p[j]] == j) // acrescenta 0 se vier da direita (f2)
				nz_sig[it]++;         // ou 1 se vier da esquerda (f1)
			j = nz_p[j];
		}
		quadratic_probing_insert(nz_hsig, nz_hval, nz_sig[it], it, hnnos);
		nz_dr[it] = y;
	}

	__syncthreads(); // espera todas as threads chegarem até aqui

	if (i < (nfol-1)) { // nos internos calculam distancia ate a raiz e armazena o caminho
						// (assinatura em bits) até o raiz
		y = 0;
		j = it+nfol+1;  // associa threads com os nos internos
		nz_sig[j] = 1;
		if (nz_p[j] == -1) j = -1;
		while (j != -1) {
			y = y + nz_br[j]; // acumula a distancia
			if (nz_p[j] == -1) 
				break;
			nz_sig[it+nfol+1] <<= 1; // acumula o caminho
			if (nz_f1[nz_p[j]] == j)    // acrescenta 0 se vier da direita (f2)
				nz_sig[it+nfol+1]++; // ou 1 se vier da esquerda (f1)
			j = nz_p[j];
		}
		quadratic_probing_insert(nz_hsig, nz_hval, nz_sig[it+nfol+1], (it+nfol+1), hnnos);
		nz_dr[it+nfol+1] = y;
	}
	
	__syncthreads(); // espera todas as threads chegarem até aqui

	// se nfol (numero de especies) for impar, usamos nfol threads
	// se nfol (numero de especies) for par, usamos nfol-1 threads
	// isso evita termos que tratar de elementos restantes
	
	if ( (nfol % 2) == 0) {
		nthreads = nfol - 1; // nfol é par: cada thread calcula nfol/2 distancias 
		a = nfol / 2; // quantidade de elementos por thread
	} else {
		nthreads = nfol;	 // nfol é ímpar: cada thread calcula (nfol-1)/2 distancias
		a = (nfol - 1) / 2;  // quantidade de elementos por thread
	}

	if (i < nthreads) {
		for( b = i*a; b < a+(i*a); b++) {
			r = row_index(b, nfol);
			c = column_index(b, nfol);
			sig1 = nz_sig[r+ib*nnos];
			sig2 = nz_sig[c+ib*nnos];
			sig3 = 1; // inicia com 1 para diferenciar das demais assinaturas, i.e., 10, 100 etc
			bits = 0; // conta quantos bits sao iguais
			sig4 = 1; // recebe assinatura invertida
			while ( (sig1 & 1) == (sig2 & 1) && bits < 32) { // compara bit menos significativo
				bit = (sig1 & 1);
				bits++;
				sig1 >>= 1; // avanca para proximo bit
				sig2 >>= 1; // avanca para proximo bit
				sig3 <<= 1; // armazena bits coincidentes - caminho do ancestral comum
				if (bit) 
					sig3++; // soma 1 ou 0
			}
			while (bits>0) { // inverte a assinatura coincidente incluindo um 1 mais a esquerda
				sig4 <<= 1;
				if (sig3 & 1)
					sig4++;
				sig3 >>= 1;
				bits--;
			}
			ancc = quadratic_probing_search(nz_hsig, nz_hval, sig4, hnnos);
			nz_de[b+ib*ennos] = nz_dr[r+ib*nnos] + nz_dr[c+ib*nnos] - 2*nz_dr[ancc];
		}
	}
}
