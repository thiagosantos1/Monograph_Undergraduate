#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

#define TRUE 1

unsigned int EMPTY = UINT_MAX;

char str[200];
int i, j, k, e;
FILE *fp;
int line=1;
int nnos, idx_ni, nfol; // numero de nos, indice de nos internos, numero de folhas
int hnnos; // tamanho da tabela hash
int ennos; // tamanho do vetor com as distancias entre as especies (matriz triangular superior)
int pos_ins, n_ins; // posicao de insercoes e numero de insercoes
int *nz; // contem indice do no; para os nos a serem inseridos, contem o indice do ponto de insercao
		 // para os nos internos a serem usados na insercao, contem -2
float *nz_br; // distancia do ramo (branch)
float *nz_dr; // distancias ate o no raiz
float *nz_de; // distancias entre especies
int *nz_qf; // altura do no
int *nz_qe; // quantidade de especies abaixo do no
int *nz_p; // pai do no
int *nz_f1; // filho da esquerda do no
int *nz_f2; // filho da direita do no
float *nz_trait; //característica a ser comparada com cada espécie 
float *nz_class_range; //Faixa para as classe de distância
float *nz_class_value; //Valores medios de I de Moran por classe de distância
float *nz_class_media; //Valores medios de I de Moran por classe de distância
float *nz_class_variance; //Variancia para cada classe de distância
unsigned int *nz_sig; // assinatura do no - da o caminho em bits ate o raiz
unsigned int *nz_hsig; // hash da assinatura do no
unsigned int *nz_hval; // indice do no na tabela hash
long long GPU_start_time;
long long GPU_time;

// pointers to GPU memory
int *nz_d;
float *nz_br_d;
float *nz_dr_d;
float *nz_de_d;
int *nz_qf_d;
int *nz_qe_d;
int *nz_p_d;
int *nz_f1_d;
int *nz_f2_d;
float *nz_trait_d; 
float *nz_class_range_d; 
float *nz_class_value_d; 
unsigned int *nz_sig_d;
unsigned int *nz_hsig_d;
unsigned int *nz_hval_d;
//int pos_ins_d, idx_ni_d;


//
char *symb, **nz_sy;
char str_tmp[100];
char str_float[30];
int nbint, nbuint, nbhuint, nbfloat, nbefloat; // tamanho em bytes dos tipos basicos
hiprandState *seed_d;
float zero = 0.0; // para facilitar impressao da matriz de distancias


// Forward function declarations
long long start_timer();
long long stop_timer(long long start_time, char *name);

// print tree in newick format
char *toNewick(int raiz, int base);

// find next prime number greater than n
int nextprime( int n );

// kernel
__global__ void Load_memory_global_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2);
__global__ void My_Load_memory_global_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2);

__global__ void Load_memory_shared_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2);

__global__ void Insert_tree_Gpu(int nnos, int hnnos, int pos_ins, int idx_ni, int *nz, float *nz_br, int *nz_qf, int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, hiprandState *states, unsigned long seed);

__global__ void Matrix_distance_Gpu(int nnos, int hnnos, int *nz, float *nz_br, float *nz_dr, float *nz_de, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, unsigned int *nz_sig, unsigned int *nz_hsig, unsigned int *nz_hval);


__global__ void I_moran_Gpu(int nnos, int nrClass, float *nz_de, float *nz_trait, float *nz_class_range, float *nz_class_value, float MeanY, float Variance);

// auxiliary kernel functions
__device__ int quadratic_probing_insert(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int val, int hnnos);
__device__ int quadratic_probing_search(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int hnnos);

__device__ inline void atomicFloatAdd(float *address, float val);

// Main program
int main(int argc, char *argv[])
{
	int qtdArvores = 1;
	int qtdBlock = 1;
	int qtdThreadsPerBlock = 1;
	int qualArvore = 0;	
	int tipoTransferencia = 1;
	long long *vetorTempo;
	
	printf("\nSyntax: newick <#qtdBlocos #qtdThreads <#numeroArvoreImprimir <#tipoTransferencia >>>");
	printf("\n   tipoTransferencia:");
	printf("\n       1: replicação feita na GPU, utilizando a memória GLOBAL como origem da copia");
	printf("\n       2: replicação feita na GPU, utilizando a memória COMPARTILHADA para acelerar transferência");
	printf("\n       3: replicação feita na CPU, em seguida, todos os dados são copiados para a GPU\n\n");



	if (argc >= 2)
		sscanf(argv[1], "%d", &qtdBlock);
	if (argc >= 3)
		sscanf(argv[2], "%d", &qtdThreadsPerBlock);
	if (argc >= 4)
		sscanf(argv[3], "%d", &qualArvore);
	if (argc >= 5)
		sscanf(argv[4], "%d", &tipoTransferencia);

	printf("qtdBlock: %d qtdThreadsPerBlock: %d => qtdArvores: %d) \n", qtdBlock, qtdThreadsPerBlock, qtdBlock*qtdThreadsPerBlock);
	printf("qualArvore: %d\n", qualArvore);
	
	vetorTempo = (long long *) malloc(5 * sizeof(long long));
	GPU_start_time = start_timer();
	dim3 grid(qtdBlock), block(qtdThreadsPerBlock);
	//Total de threads a serem criadas, conforme a quantidade de blocos e threads por bloco
	qtdArvores = grid.x * block.x;
 
	printf("qtdArvores  %d. \n", qtdArvores);
	fp = fopen("wellParser.out", "r");
	if (fp == NULL) {
		printf("\nCannot open file\n");
		exit(0);
	}

	fscanf(fp,"%d %d", &nnos, &idx_ni);
	printf("No nos: %d, Indice no interno: %d\n", nnos, idx_ni);

	nfol = nnos / 2;

	fscanf(fp,"%d %d", &pos_ins, &n_ins);
	printf("Inserir %d especies a partir de %d\n", n_ins, pos_ins);

	printf("Arvore: ");
	nz = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz[i]);
		printf("%d ", nz[i]);
	}
	printf("\n");

	printf("Simbolos: ");
	symb = (char *) malloc(50);
	nz_sy = (char **) malloc(nnos * sizeof(char *));
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%s", symb);
		nz_sy[i] = (char *) malloc(50);
		strcpy(nz_sy[i], symb);
		printf("%s ", nz_sy[i]);
	}
	printf("\n");

	nz_dr = (float *) malloc(nnos * sizeof(float) * qtdArvores);
	
	ennos = (nfol * (nfol - 1)) / 2;
	size_t tamanho_ui = (unsigned int) (ennos * sizeof(float) * qtdArvores) ;
	nz_de = (float *) malloc( tamanho_ui );
	printf("\n\nennos * sizeof(float) * qtdArvores(%d): %u\n\n", qtdArvores, tamanho_ui );

	printf("Ramos: ");
	nz_br = (float *) malloc(nnos * sizeof(float) * qtdArvores);

	for(i=0; i<nnos; i++) {
		fscanf(fp,"%f", &nz_br[i]);
		printf("%.2f ", nz_br[i]);
	}
	printf("\n");

	printf("No Filhos: ");
	nz_qf = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz_qf[i]);
		printf("%d ", nz_qf[i]);
	}
	printf("\n");
	
	printf("No Especies: ");
	nz_qe = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz_qe[i]);
		printf("%d ", nz_qe[i]);
	}
	printf("\n");

	printf("Pais: ");
	nz_p = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz_p[i]);
		printf("%d ", nz_p[i]);
	}
	printf("\n");

	printf("Filhos 1: ");
	nz_f1 = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz_f1[i]);
		printf("%d ", nz_f1[i]);
	}
	printf("\n");

	printf("Filhos 2: ");
	nz_f2 = (int *) malloc(nnos * sizeof(int) * qtdArvores);
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%d", &nz_f2[i]);
		printf("%d ", nz_f2[i]);
	}
	printf("\n");
	
	printf("Traits: ");
	nz_trait = (float *) malloc(nnos * sizeof(float) );
	for(i=0; i<nnos; i++) {
		fscanf(fp,"%f", &nz_trait[i]);
		printf("%f ", nz_trait[i]);
	}
	printf("\n");
	
	hnnos = nextprime(2*nnos);
	printf("\nPrimo/hnnos = %d, qtdArvores = %d, hnnos*qtdArvores = %d\n", hnnos, qtdArvores, hnnos * qtdArvores);
	nz_sig = (unsigned int *) malloc(hnnos * sizeof(unsigned int)  * qtdArvores);
	for(i=0; i<(hnnos * qtdArvores); i++) {
		nz_sig[i] = 0;
	}
	
	nz_hsig = (unsigned int *) malloc(hnnos * sizeof(unsigned int) * qtdArvores);
	nz_hval = (unsigned int *) malloc(hnnos * sizeof(unsigned int) * qtdArvores);
	
	for(i=0; i<(hnnos * qtdArvores); i++) {
		nz_hsig[i] = (unsigned int) EMPTY;
		nz_hval[i] = (unsigned int) EMPTY;
	}

	fclose(fp);

	toNewick(nnos-1, 0);
	printf(";\n");
	
	// move data to GPU
	nbint = nnos * sizeof(int);
	nbuint = nnos * sizeof(unsigned int);
	nbhuint = hnnos * sizeof(unsigned int);
	nbfloat = nnos * sizeof(float);
	nbefloat = ennos * sizeof(float);
	GPU_time = stop_timer(GPU_start_time, "\t Tempo: Preencher dados nas estruturas da CPU");


	GPU_start_time = start_timer();
	//cudaMalloc((void **)&pos_ins_d, sizeof(int));
	//cudaMalloc((void **)&idx_ni_d, sizeof(int));
/*	:printf("\n ________ \t tipos \t\t tipos*qtdArvores, nnos %d ", nnos);
	printf("\n nbint    \t %d \t\t %d ", nbint, nbint * qtdArvores);
	printf("\n nbfloat  \t %d \t\t %d ", nbfloat, nbfloat * qtdArvores);
	printf("\n nbefloat \t %d \t\t %d ", nbefloat, nbefloat * qtdArvores);
	printf("\n nbuint   \t %d \t\t %d ", nbuint, nbuint * qtdArvores);
	printf("\n ");
*/

	hipDeviceReset();

	printf("\ncurandState: %d\n", sizeof(hiprandState));

	hipMalloc((void **)&nz_d, nbint * qtdArvores);
    	hipMalloc((void **)&nz_br_d, nbfloat * qtdArvores);
    	hipMalloc((void **)&nz_dr_d, nbfloat * qtdArvores);
    	hipMalloc((void **)&nz_qf_d, nbint * qtdArvores);
    	hipMalloc((void **)&nz_qe_d, nbint * qtdArvores);
    	hipMalloc((void **)&nz_p_d, nbint * qtdArvores);
    	hipMalloc((void **)&nz_f1_d, nbint * qtdArvores);
    	hipMalloc((void **)&nz_f2_d, nbint * qtdArvores);
    	hipMalloc((void **)&seed_d, nnos*sizeof(hiprandState)*qtdArvores);
	GPU_time = stop_timer(GPU_start_time, "\t Tempo: Alocar memória na GPU");

	if( nz_d==0 ) {
		printf("couldn't allocate memory nz_d\n"); 
		return 1;
   	}
 	if( nz_br_d==0 ) {
		printf("couldn't allocate memory nz_br_d\n"); 
		return 1;
   	}
 	if( nz_dr_d==0 ) {
		printf("couldn't allocate memory nz_dr_d\n"); 
		return 1;
   	}
	if( nz_qf_d==0  ) {
		printf("couldn't allocate memory nz_qf_d\n"); 
		return 1;
   	}
 	if( nz_qe_d==0 ) {
		printf("couldn't allocate memory nz_qe_d\n"); 
		return 1;
   	} 
	if( nz_p_d==0 || nz_f1_d==0 || nz_f2_d==0 ) {
		printf("couldn't allocate memory 2\n"); 
		return 1;
   	} 
	if(seed_d ==0 ) {
		printf("couldn't allocate memory seed_d\n"); 
		return 1;
   	}


	GPU_start_time = start_timer();
	if (tipoTransferencia == 1 || tipoTransferencia == 2){
		hipMemcpy(nz_d, nz, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_br_d, nz_br, nbfloat, hipMemcpyHostToDevice);
		hipMemcpy(nz_qf_d, nz_qf, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_qe_d, nz_qe, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_p_d, nz_p, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_f1_d, nz_f1, nbint, hipMemcpyHostToDevice);
		hipMemcpy(nz_f2_d, nz_f2, nbint, hipMemcpyHostToDevice);

/*

		//cudaMemcpy(pos_ins_d, pos_ins, sizeof(int), cudaMemcpyHostToDevice);
		//cudaMemcpy(idx_ni_d, idx_ni, sizeof(int), cudaMemcpyHostToDevice);
*/
	}

	GPU_time = stop_timer(GPU_start_time, "\t Tempo para copiar dados (bases) para memória");
	vetorTempo[0] = GPU_time;
	int aux = sizeof(int)*nnos;

	/* OPÇÕES PARA GERAR OS DADOS NA GPU:
		1. Copiar os elementos das estruturas e replica-los na gpu, utilizando memória GLOBAL
		2. Copiar os elementos das estruturas e replica-los na gpu, utilizando memória COMPARTILHADA
		3. Replicar os elementos na CPU e copia-los para A GPU (memória global)
	*/
	if (tipoTransferencia == 1) {
		GPU_start_time = start_timer();
//		Load_memory_global_Gpu<<<grid, block, aux>>>(nnos, nz_d, nz_br_d, nz_dr_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d);
		My_Load_memory_global_Gpu<<<qtdArvores, nnos>>>(nnos, nz_d, nz_br_d, nz_dr_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d);
		hipDeviceSynchronize();
		GPU_time = stop_timer(GPU_start_time, "\t Tempo para copiar memória GPU (transferencia via memoria global)");
		vetorTempo[0] += GPU_time;
	}else{
		if (tipoTransferencia == 2){
			GPU_start_time = start_timer();
			Load_memory_shared_Gpu<<<grid, block, aux>>>(nnos, nz_d, nz_br_d, nz_dr_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d);
			hipDeviceSynchronize();
			GPU_time = stop_timer(GPU_start_time, "\t Tempo para copiar memória GPU (transferencia via memoria compartilhada)");
			vetorTempo[0] += GPU_time;
		}else{
				GPU_start_time = start_timer();
				int base = 0;
				for(i = 0; i < qtdArvores; i++) {
					if (i > 0) {
						for(j = 0; j < nnos; j++) {
							base = i * nnos;
							nz[base+j] = nz[j] + (nz[j] >= 0 ? base : 0);
							nz_br[base+j] = nz_br[j];
							nz_dr[base+j] = 0;
							//nz_de[base+j] = nz_de[j];
							nz_qf[base+j] = nz_qf[j];
							nz_qe[base+j] = nz_qe[j];
							nz_p[base+j] = nz_p[j] + (nz_p[j] >= 0 ? base : 0);
							nz_f1[base+j] = nz_f1[j] + (nz_f1[j] >= 0 ? base : 0);
							nz_f2[base+j] = nz_f2[j] + (nz_f2[j] >= 0 ? base : 0);
						}
						nz[nfol] = -i;
					}
				}
				hipMemcpy(nz_d, nz, nbint * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_br_d, nz_br, nbfloat * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_dr_d, nz_dr, nbfloat * qtdArvores, hipMemcpyHostToDevice);
				//cudaMemcpy(nz_de_d, nz_de, nbefloat * qtdArvores, cudaMemcpyHostToDevice);
				hipMemcpy(nz_qf_d, nz_qf, nbint * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_qe_d, nz_qe, nbint * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_p_d, nz_p, nbint * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_f1_d, nz_f1, nbint * qtdArvores, hipMemcpyHostToDevice);
				hipMemcpy(nz_f2_d, nz_f2, nbint * qtdArvores, hipMemcpyHostToDevice);

				GPU_time = stop_timer(GPU_start_time, "\t Tempo para copiar da CPU->GPU (carregar dados)");
				vetorTempo[0] += GPU_time;
		}
	}







	hipDeviceSynchronize();
	/**************************************************
	*
	* I N S E R I R   E S P E C I E S   P E R D I D A S 
	*
	******************************************************/
	// call kernel

	GPU_start_time = start_timer();
	if (n_ins > 0){ //se houver nós a inserir
		Insert_tree_Gpu<<<grid, block, aux>>>(nnos, hnnos, pos_ins, idx_ni, nz_d, nz_br_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d, seed_d, time(NULL));
		printf("Erro (inserir): %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	hipDeviceSynchronize();
	GPU_time = stop_timer(GPU_start_time, "\t Tempo para incluir nós na árvore");
	vetorTempo[1] = GPU_time;


				
	//alocar memoria para outras vetores

    	hipMalloc((void **)&nz_de_d, nbefloat * qtdArvores);
    	hipMalloc((void **)&nz_sig_d, nbhuint * qtdArvores);
    	hipMalloc((void **)&nz_hsig_d, nbhuint * qtdArvores);
    	hipMalloc((void **)&nz_hval_d, nbhuint * qtdArvores);
 	if( nz_de_d==0  ) {
		printf("couldn't allocate memory nz_de_d\n"); 
		return 1;
   	} 
	if( nz_sig_d==0) {
		printf("couldn't allocate memory nz_sig_d\n"); 
		return 1;
   	} 
	if( nz_hsig_d==0 ) {
		printf("couldn't allocate memory nz_hsig_d\n"); 
		return 1;
   	} 
	if( nz_hval_d==0 ) {
		printf("couldn't allocate memory nz_hval_d\n"); 
		return 1;
   	} 

	hipMemcpy(nz_sig_d, nz_sig, nbhuint * qtdArvores, hipMemcpyHostToDevice);
	hipMemcpy(nz_hsig_d, nz_hsig, nbhuint * qtdArvores, hipMemcpyHostToDevice);
	hipMemcpy(nz_hval_d, nz_hval, nbhuint * qtdArvores, hipMemcpyHostToDevice);

	/**************************************************
	*
	* C A L C U L A R   A   M A T R I Z   D E   D I S T A N C I A 
	*
	******************************************************/
	hipDeviceSynchronize();
	GPU_start_time = start_timer();
//	int nb = qtdArvores;
	Matrix_distance_Gpu<<<qtdArvores, nfol>>>(nnos, hnnos, nz_d, nz_br_d, nz_dr_d, nz_de_d, nz_qf_d, nz_qe_d, nz_p_d, nz_f1_d, nz_f2_d, nz_sig_d, nz_hsig_d, nz_hval_d);
	hipDeviceSynchronize();
	printf("Erro (matrix distancia): %s\n", hipGetErrorString( hipGetLastError() ) );
	GPU_time = stop_timer(GPU_start_time, "\t Tempo total para calcular a matriz de distância");

	vetorTempo[2] = GPU_time;

	GPU_start_time = start_timer();

	// copy data back to the CPU
	//cudaMemcpy(pos_ins, pos_ins_d, sizeof(int), cudaMemcpyDeviceToHost);
	//cudaMemcpy(idx_ni, idx_ni_d, sizeof(int), cudaMemcpyDeviceToHost);
	hipMemcpy(nz, nz_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_br, nz_br_d, nbfloat * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_dr, nz_dr_d, nbfloat * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_de, nz_de_d, nbefloat * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_qf, nz_qf_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_qe, nz_qe_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_p, nz_p_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_f1, nz_f1_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_f2, nz_f2_d, nbint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_sig, nz_sig_d, nbhuint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_hsig, nz_hsig_d, nbhuint * qtdArvores, hipMemcpyDeviceToHost);
	hipMemcpy(nz_hval, nz_hval_d, nbhuint * qtdArvores , hipMemcpyDeviceToHost);
	GPU_time = stop_timer(GPU_start_time, "\t Tempo copiar dados de volta (GPU -> cpu): ");
	vetorTempo[3] = GPU_time; //Copiar dados da Gpu para cpu
	
/*
	printf("\n\nImprimir uma arvore: \n");
	toNewick(nnos-1, 0);
	printf(";\n");
*/

	hipDeviceSynchronize();
	//Desalocar memoria da GPU para utilizar no próximo kernel
	GPU_start_time = start_timer();

	hipFree(nz_d);    
	hipFree(nz_br_d);
	hipFree(nz_dr_d);
	hipFree(nz_qf_d);
	hipFree(nz_qe_d);
	hipFree(nz_p_d);    
	hipFree(nz_f1_d);
	hipFree(nz_f2_d);

	hipFree(nz_sig_d);
	hipFree(nz_hsig_d);
	hipFree(nz_hval_d);
	hipFree(seed_d);

	free(nz_qf);
	free(nz_qe);
	free(nz_f1);
	free(nz_f2);
	free(nz_p);
	free(nz_dr);

	free(nz_sig);
	free(nz_hsig);
	free(nz_hval);

	GPU_time = stop_timer(GPU_start_time, "\t Tempo: Liberar memória GPU");


	/**************************************************
	*
	* C A L C U L A R   I   D E   M O R A N
	*
	******************************************************/
	//Aloca posicoes em memoria para armazenar as classes de distância
	int nrClass = 4;
	float maiorDistancia=0, menorDistancia = nz_de[0], salto;
	nz_class_range = (float *) malloc((nrClass+1) * sizeof(float));
	nz_class_value = (float *) malloc(nrClass * sizeof(float) * qtdArvores);
	nz_class_media = (float *) malloc(nrClass * sizeof(float) );
	nz_class_variance = (float *) malloc(nrClass * sizeof(float) );
	//As classes são definidas de forma igual, entre o maior e menor valor	
	for (i=0;i<ennos;i++){
		if (maiorDistancia < nz_de[i])
			maiorDistancia = nz_de[i];	
		if (menorDistancia > nz_de[i])
			menorDistancia = nz_de[i];
	}

	//nz_class_range[0] = menorDistancia;
	salto = (maiorDistancia - menorDistancia)/nrClass;
	for(i=0;i<nrClass;i++){
		nz_class_range[i] = menorDistancia;
		nz_class_value[i] = 0.0;
		menorDistancia += salto;
	}
	nz_class_range[0] -= nz_class_range[0]/2; //para incluir distancias iguais ao menor valor
	nz_class_range[i] = maiorDistancia;

	//realiza uma cópia do vetor de características (são as mesmas para todas as especies, independente da posição na árvore)
    	hipMalloc((void **)&nz_trait_d, nbfloat);
    	hipMalloc((void **)&nz_class_range_d, sizeof(float) * (nrClass+1)); //+1 para guardar a faixa final da classe
    	hipMalloc((void **)&nz_class_value_d, sizeof(float) * nrClass * qtdArvores);

	hipMemcpy(nz_trait_d, nz_trait, nbfloat, hipMemcpyHostToDevice);
	hipMemcpy(nz_class_range_d, nz_class_range, sizeof(float) * (nrClass+1), hipMemcpyHostToDevice);
	hipMemcpy(nz_class_value_d, nz_class_value, sizeof(float) * nrClass * qtdArvores, hipMemcpyHostToDevice);



	float Variance, MeanY, SumW;

	SumW = 0;
  	Variance = 0;

  	for (int d=0;d<nfol;d++){
    		SumW = SumW + nz_trait[d];
    		Variance = Variance + pow(nz_trait[d],2);
	}
  	MeanY = SumW / nfol;
	Variance = Variance - (pow(SumW, 2) / nfol);

	hipDeviceSynchronize();
	GPU_start_time = start_timer();
	aux = sizeof(float)*(nrClass+1);
	I_moran_Gpu<<<qtdArvores, nfol, aux>>>(nnos, nrClass, nz_de_d, nz_trait_d, nz_class_range_d, nz_class_value_d, MeanY, Variance);
	hipDeviceSynchronize();
	printf("Erro (I_moran_Gpu): %s\n", hipGetErrorString( hipGetLastError() ) );
	GPU_time = stop_timer(GPU_start_time, "\t Tempo para calcular o Indice de Moran): ");
	vetorTempo[4] = GPU_time; //Copiar dados da Gpu para cpu


	//Traz os resultados de volta (GPU para Host), as medias são armazenadas no início do vetor
	hipMemcpy(nz_class_value, nz_class_value_d, nrClass * sizeof(float) * qtdArvores, hipMemcpyDeviceToHost);

	//Calcula a media por classe e a variancia
	float media;
	int nrArvore;
	for(i=0;i<nrClass;i++){
		media = 0;
		for (nrArvore=i;nrArvore<(qtdArvores*nrClass);nrArvore+=nrClass){
			media += nz_class_value[nrArvore];
		}
		nz_class_media[i] = media / qtdArvores;
	}
	//calculo da variancia	
	for(i=0;i<nrClass;i++){
		media = 0;
		for (nrArvore=i;nrArvore<(qtdArvores*nrClass);nrArvore+=nrClass){
			media += pow((nz_class_value[nrArvore] -  nz_class_media[i]), 2);
		}
		nz_class_variance[i] = media / qtdArvores;
	}

	GPU_start_time = start_timer();


	/**************************************************
	*
	* E X I B I R   R E S U L T A D O S 
	*
	******************************************************/

/*

	printf("\nnz_sy, ");
	for (int jx=0;jx<qtdArvores;jx++)	
		for(i=0;i<nnos;i++){
			printf("%s,", nz_sy[i]);
		}

	printf("\nnz, ");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz[i]);
	}
*/
	printf("\nnz_br,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%f,", nz_br[i]);
		if (i == 10000) //limitar impressao para nao deixar os arquivos muito grandes
			break;
	}
/*
	printf("\nnz_dr,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%f,", nz_dr[i]);
	}

	printf("\nnz_qf,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz_qf[i]);
	}


	printf("\nnz_qe,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz_qe[i]);
	}


	printf("\nnz_p,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz_p[i]);
//		if (i == 10000)
//			break;
	}

	printf("\nnz_f1,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz_f1[i]);
		if (i == 10000)
			break;
	}

	printf("\nnz_f2,");
	for(i=0;i<(qtdArvores*nnos);i++){
		printf("%d,", nz_f2[i]);
		if (i == 10000)
			break;
	}
*/
	printf("\nnz_class,");
	for(i=0;i<(nrClass);i++){
		printf("\t\n [%d] %f => value: %f ; media: %f ; variance: %f ", i, nz_class_range[i], nz_class_value[i], nz_class_media[i], nz_class_variance[i]);
	}

	printf("\n");
	printf("\n");

/*
	for(i=1;i<=qtdArvores;i++){
		//toNewick((nnos)-1);
		toNewick((i*nnos)-1, (nnos*(i-1)));
		printf(";\n");
	}

	printf("Pais: ");
	for(i=0; i<(nnos*qtdArvores); i++) {
		printf("%d ", nz_p[i]);
	}
	printf("\n");

	printf("Dst Raiz: ");
	for(i=0; i<(nnos*qtdArvores); i++) {
		if ((i-((i/nnos)*nnos)) == nfol) continue; // desconta o no da posicao nfol
		printf("%.2f ", nz_dr[i]); // pois este nao e usado
	}
	printf("\n");



	printf("Assinatura: ");
	for(i=0; (i<(hnnos * qtdArvores)); i++) {
		if ((i-((i/nnos)*nnos)) == nfol) continue; // desconta o no da posicao nfol
		if (i == nfol) continue;
		printf("%u ", nz_sig[i]);
	}
	printf("\n");
	
	printf("Hash Sign: ");
	for(i=0; (i<(hnnos * qtdArvores)); i++) {
		if (i == nfol) continue;
		if ((i-((i/nnos)*nnos)) == nfol) continue; // desconta o no da posicao nfol
		printf("%u ", nz_hsig[i]);
	}
	printf("\n");
	
	printf("Hash Val: ");
	for(i=0; (i<(hnnos * qtdArvores)); i++) {
		if ((i-((i/nnos)*nnos)) == nfol) continue; // desconta o no da posicao nfol
		if (i == nfol) continue;
		printf("%u ", nz_hval[i]);
	}
	printf("\n");
*/

	e = 0; // indexa a matriz triangular superior (representada num array) que contem a distancia
		   // entre as especies
	printf("Distancias: \n");
	printf("%7s ", nz_sy[0]);


	for(i=1; i<nfol; i++)
		printf("%4s ", nz_sy[i]);
	printf("\n");
	for(i=0; i<nfol; i++) {
		printf("%3s ", nz_sy[i]);
//		if (i >= (nfol-3)){
			for(j=0; j<=i; j++)
				printf("%.2f ", zero);
			for(k=i+1; k<nfol; k++) {
					printf("%.2f ", nz_de[e+(qualArvore*ennos)]);
				e++;
			}
//		}
		printf("\n");
	}

/*
	printf("\n\nnz_de: ");
	for(k=0; k<(ennos*qtdArvores); k++) {
			printf("%.2f ", nz_de[k]);
	}	
*/
	GPU_time = stop_timer(GPU_start_time, "\t Tempo mostrar dados em tela");

	GPU_start_time = start_timer();
	printf("\n");
	free(nz);
	free(nz_br);
	free(nz_de);
	free(symb);
	free(nz_sy);
	GPU_time = stop_timer(GPU_start_time, "\t Tempo: Liberar memoria da CPU");

	GPU_start_time = start_timer();
	hipFree(nz_de_d);
	GPU_time = stop_timer(GPU_start_time, "\t Tempo: Liberar memória GPU (matriz de distancia)");

	printf("\n\n===================   R e s u m o    d o s    T e m p o s   =======================");

	printf("\nCPU -> GPU\tIncluir Esp. \tMatriz dist.\tGPU-> CPU\tI de Moran (em sec)");
	printf("\n%.5f \t", ((float) vetorTempo[0]) / (1000 * 1000));
	printf("%.5f \t", ((float) vetorTempo[1]) / (1000 * 1000));
	printf("%.5f \t", ((float) vetorTempo[2]) / (1000 * 1000));
	printf("%.5f \t", ((float) vetorTempo[3]) / (1000 * 1000));
	printf("%.5f \t\n", ((float) vetorTempo[4]) / (1000 * 1000));
	
	free(vetorTempo);
	return 0;
}


// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}



char *toNewick(int idRaiz, int base) {
    strcpy(str_tmp,"");
    strcpy(str_float,"");
	if (nz_f1[idRaiz] < 0) { // Não tem filhos 
		if ((idRaiz-base) < 0 || (idRaiz-base) > (nnos-1))
			printf("ERRO %d\n", (idRaiz-base));
		else
			strcat (str_tmp, nz_sy[idRaiz-base]);
		strcat (str_tmp, ":");
		sprintf(str_float,"%0.2f", nz_br[idRaiz]); 
		strcat (str_tmp, str_float);
		return str_tmp;
	} else { // Tem filhos 
		printf("(");
		printf("%s", toNewick(nz_f1[idRaiz], base));
		printf(",");
		printf("%s", toNewick(nz_f2[idRaiz], base));
		printf(")");
		printf("%s", nz_sy[idRaiz-base]);
		printf(":");
		sprintf(str_float,"%0.2f", nz_br[idRaiz]); 
		printf("%s", str_float);
		return "";
	}
}

int nextprime( int n ) {
	int Divisor, PossiblePrime;
	int FoundPrime;

	PossiblePrime = n;
	if( PossiblePrime <= 2 )
		PossiblePrime = 2;
	else
		if( PossiblePrime != 3 ) {
			if( PossiblePrime % 2 == 0 )
				PossiblePrime++;	/* Need An Odd Number */
			for( ; ; PossiblePrime += 2 ) {
				FoundPrime = !TRUE;
				for( Divisor = 3; PossiblePrime % Divisor; Divisor += 2 )
					if( Divisor * Divisor > PossiblePrime ) {
						FoundPrime = TRUE;
						break;
					}
				if( FoundPrime )
					break;
			}
		}
	return PossiblePrime;
}

__device__ int quadratic_probing_insert(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int val, int hnnos) {
    unsigned int j, hk, old;
    
    int ib = blockIdx.x; // identificador do bloco
    j = 0;
    hk = sig  % hnnos;
    while(j < hnnos) {
    	old = atomicCAS(&nz_hsig[hk+ib*hnnos], UINT_MAX, sig); // se posicao estiver vazia (UINT_MAX = EMPTY)
		if (old == UINT_MAX) {
			nz_hval[hk+ib*hnnos] = val;
			return (hk+ib*hnnos);
    	}
        j++;
        hk = (hk + j * j) % hnnos;
//        hk = (hk + j) % hnnos;
    }
    return (-1);
}

__device__ int quadratic_probing_search(unsigned int *nz_hsig, unsigned int *nz_hval, unsigned int sig, int hnnos) {
    unsigned int j, hk;
    
    int ib = blockIdx.x; // identificador do bloco
    j = 0;
    hk = sig  % hnnos;
    while(j < hnnos) {
		if (nz_hsig[hk+ib*hnnos] == sig) {
			return (nz_hval[hk+ib*hnnos]);
    	}
        j++;
        hk = (hk + j * j) % hnnos;
//        hk = (hk + j) % hnnos;
    }
    return (-1);
}

// estas duas funcoes sao usada para mapear os indices de um array para uma matriz triangular 
// superior correspondente (sem a diagonal). para uma matriz nxn, o array terá n(n-1)/2 elementos

__host__ __device__ int row_index( int i, int M ){ // retorna o indice da linha
	M--;
    float m = M;
    float row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(float)i - 7) )) / -2;
    if( row == (float)(int) row ) row -= 1;
    return (int) row;
}

__host__ __device__ int column_index( int i, int M ){ // retorna o indice da coluna
    int row = row_index( i, M);
    M--;
    return 1 + (i - M * row + row*(row+1) / 2);
}


__global__ void Load_memory_global_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2) {
	int i = threadIdx.x; // identificador da thread
	int index, new_index;
	int base = (blockIdx.x * blockDim.x * nnos) + nnos * i;

	//Todas as threads copiam os dados para suas respectivas áreas
	for(index = 0; index < nnos; index++){
		new_index = base+index;		
		nz[new_index] = nz[index] + (nz[index] >= 0 ? base : 0);
		nz_br[new_index] = nz_br[index];
		nz_dr[new_index] = 0;
		nz_qf[new_index] = nz_qf[index];
		nz_qe[new_index] = nz_qe[index];
		nz_p[new_index] = nz_p[index] + (nz_p[index] >= 0 ? base : 0);
		nz_f1[new_index] = nz_f1[index] + (nz_f1[index] >= 0 ? base : 0);
		nz_f2[new_index] = nz_f2[index] + (nz_f2[index] >= 0 ? base : 0);
	}

}

__global__ void My_Load_memory_global_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2) {
	int index = threadIdx.x; // identificador da thread
	int new_index;
	int base = (blockIdx.x * blockDim.x);

	//Todas as threads copiam os dados para suas respectivas áreas
		new_index = base+index;		
		nz[new_index] = nz[index] + (nz[index] >= 0 ? base : 0);
		nz_br[new_index] = nz_br[index];
		nz_dr[new_index] = 0;
		nz_qf[new_index] = nz_qf[index];
		nz_qe[new_index] = nz_qe[index];
		nz_p[new_index] = nz_p[index] + (nz_p[index] >= 0 ? base : 0);
		nz_f1[new_index] = nz_f1[index] + (nz_f1[index] >= 0 ? base : 0);
		nz_f2[new_index] = nz_f2[index] + (nz_f2[index] >= 0 ? base : 0);

}

__global__ void Load_memory_shared_Gpu(int nnos, int *nz, float *nz_br, float *nz_dr, int *nz_qf, int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2) {
	extern __shared__ float nzTemp[];

	int i = threadIdx.x; // identificador da thread
	int index;


	int base = (blockIdx.x * blockDim.x * nnos) + nnos * i;

	//Copiar dados do vetor NZ
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz[base+index] = (int) (nzTemp[index] + (nzTemp[index] >= 0 ? base : 0));
		__syncthreads(); 

	//Copiar dados do vetor BR
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_br[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz_br[base+index] = nzTemp[index];
		__syncthreads(); 

	//Copiar dados do vetor QF
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_qf[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++){
			nz_dr[base+index] = 0;			
			nz_qf[base+index] = nzTemp[index];
		}
		__syncthreads(); 

	//Copiar dados do vetor QE
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_qe[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz_qe[base+index] = nzTemp[index];
		__syncthreads(); 

	//Copiar dados do vetor P
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_p[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz_p[base+index] = (int) (nzTemp[index] + (nzTemp[index] >= 0 ? base : 0));
		__syncthreads(); 

	//Copiar dados do vetor F1
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_f1[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz_f1[base+index] = (int) (nzTemp[index] + (nzTemp[index] >= 0 ? base : 0));
		__syncthreads(); 

	//Copiar dados do vetor F2
		if (threadIdx.x == 0)
			for(index = 0; index < nnos; index++)
				nzTemp[index] = nz_f2[index];
		__syncthreads(); 

		for(index = 0; index < nnos; index++)
			nz_f2[base+index] = (int) (nzTemp[index] + (nzTemp[index] >= 0 ? base : 0));
//		__syncthreads(); 

}




__global__ void Insert_tree_Gpu(int nnos, int hnnos, int pos_ins, int idx_ni, int *nz, float *nz_br, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, hiprandState *states, unsigned long seed) {
	int i = threadIdx.x; // identificador da thread
	float x; // valor gerado aleatoriamente
	unsigned int valor2; // numero entre 1 e maximo inteiro sem sinal
	unsigned int valor1; // numero entre 1 e altura da sub-arvore
	unsigned int shift = 8*sizeof(unsigned int)-1; // bits estao na faixa 0-31, e nao em 1-32
	unsigned int mask=1<<shift; // recebe 1 deslocado 31 vezes p/ direita 
								// (10000000 00000000 00000000 00000000)
	__shared__ int nfol; // numero de folhas da arvore
	int indMdcc; // no a partir do qual sera inserido uma especie
	int indNewNode; // aponta para o no internto a ser inserido junto com a especie a ser inserida
				 // idx_ni e o indice inicial dos nos internos a serem inseridos. Este indice
				 // cresce da direita para a esquerda. Veja que pos_ins aponta para a primeira
				 // especie a ser inserida. Serao inseridas nfol-pos_ins+1 especies.
	int indSisterSpecies;	
	int index;
	int indNewSpecies;

	int base = (blockIdx.x * blockDim.x * nnos) + nnos * i;

	index = 0;
	indSisterSpecies = 0;

	nfol = nnos / 2; // folhas estao na metade inferior
	hiprand_init(seed+i, base, 0, &states[base]);  // 	Initialize CURAND
	for(indNewSpecies=(base+pos_ins);indNewSpecies < (base+nfol);indNewSpecies++){
		hiprand(&states[base]);
		x = hiprand_uniform (&states[base]);      // gera numero aleatorio
		indNewNode = base + (idx_ni - ((indNewSpecies-base) - pos_ins)); // recebe um no interno a ser usado na insercao das especies
       								  
		indMdcc = nz[indNewSpecies]; // a posicao species [pos_ins <= species < nfol] contem o indice do no interno que
				   // sera usado para inserir a especie, i.e., ponto inicial de insercao (MDCC-most derived consensus clade)
					   
		valor1 = (int) (1 + x*nz_qf[indMdcc]); // numero entre 1 e altura da sub-arvore
		valor2 = (unsigned int) (1 + x*UINT_MAX);    // numero entre 1 e maximo inteiro sem sinal

		// a insercao e feita a partir do ponto de insercao mas seguindo os bits de valor2
		// se o bit for 1 avanca para a esquerda (f1) e se for 0 avanca para a direita (f2)
		if (indMdcc <= (base+nfol)) //Se o ponto de inserção for uma folha, então sobe um nível
			indMdcc = nz_p[indMdcc];
		else
			while (valor1 > 0) { // faca enquando nao alcancar a altura do no em questao ou um no
						 // folha seja alcancado.
				if(valor2 & mask)
					if (nz_f1[indMdcc] <= (base+nfol)) break;
					else indMdcc = nz_f1[indMdcc]; // avanca para proximo filho
				else
					if (nz_f2[indMdcc] <= (base+nfol)) break;
					else indMdcc = nz_f2[indMdcc]; // avanca para proximo filho
				valor2 <<= 1; // avanca para proximo bit
				valor1--; // diminui altura da arvore
			}

		//
		// convencao: f1 aa esquerda e f2 aa direita
		//
		x = hiprand_uniform (&states[base]);      // gera numero aleatorio - reuso de x
		if(valor2 & mask) { // insere no aa direita (f2) do no folha (especie) atual (f1)
			indSisterSpecies = nz_f1[indMdcc]; //nó a partir do qual o calculo do brach para a nova especie será realizado
			nz_f1[indNewNode] = nz_f1[indMdcc];
			nz_f2[indNewNode] = indNewSpecies;
			nz_p[nz_f1[indNewNode]] = indNewNode;
			nz_f1[indMdcc] = indNewNode;
			nz_br[indNewNode] = x * nz_br[nz_f1[indNewNode]];
			nz_br[nz_f1[indNewNode]] -= nz_br[indNewNode];
			nz_qf[indNewNode] = nz_qf[nz_f1[indNewNode]]++;
			nz_qe[indNewNode] = nz_qe[nz_f1[indNewNode]]++;
		}
		else { // insere no aa esquerda (f1) do no folha (especie) atual (f2)
			indSisterSpecies = nz_f2[indMdcc]; //nó a partir do qual o calculo do brach para a nova especie será realizado
			nz_f1[indNewNode] = indNewSpecies;
			nz_f2[indNewNode] = nz_f2[indMdcc];
			nz_p[nz_f2[indNewNode]] = indNewNode;
			nz_f2[indMdcc] = indNewNode;
			//Dividir o branch do nó "quebrado", de forma proporcional para o novo nó PAI (indNewNode)
			nz_br[indNewNode] = x * nz_br[nz_f2[indNewNode]];
			nz_br[nz_f2[indNewNode]] -= nz_br[indNewNode];
			nz_qf[indNewNode] = nz_qf[nz_f2[indNewNode]]++;	
			//Atualizar informacoes de quantidade de especies
			nz_qe[indNewNode] = nz_qe[nz_f2[indNewNode]]++;
		}			
		//atualiza vetor de pais

		nz_p[indNewSpecies] = indNewNode;
		nz_p[indNewNode] = indMdcc;
		nz_qe[indNewSpecies] = 1;

		//atualizar a qtde de especies e qtd de filhos
		index = nz_p[indNewNode];
		x = nnos/2;
		while( index > -1 || x <= 0 ){
			nz_qe[index] += 1; 
//			if (nz_f1[index] == -2 || nz_f2[index] == -2) break;
			if ( (nz_f1[index] >= (base+nfol) && nz_qf[nz_f1[index]] >= nz_qf[index]) || (nz_f2[index] >= (base+nfol) && nz_qf[nz_f2[index]] >= nz_qf[index])) 
				nz_qf[index] += 1;

			index = nz_p[index]; 
			x--;

		}

		//Calcular distancia para o nó inserido
		x = hiprand_uniform (&states[base]);      // gera numero aleatorio - reuso de x
		if (indSisterSpecies < (base+nfol)) //se irma eh folha, então branch deve possuir tamanho igual a irma
			nz_br[indNewSpecies] = nz_br[indSisterSpecies];
		else {
			valor2 = (unsigned int) (1 + x*UINT_MAX);    // numero entre 1 e maximo inteiro sem sinal
			nz_br[indNewSpecies] = 0.0;
			index = indSisterSpecies;

			while (true){
				nz_br[indNewSpecies] += nz_br[index];
				if (valor2 & mask){
					if (nz_f1[index] == -2) break;
					index = nz_f1[index];
				}else{
					if (nz_f2[index] == -2) break;
					index = nz_f2[index];

				}
			}

		}
	
	}
}



__global__ void Matrix_distance_Gpu(int nnos, int hnnos, int *nz, float *nz_br, float *nz_dr, float *nz_de, int *nz_qf,int *nz_qe, int *nz_p, int *nz_f1, int *nz_f2, unsigned int *nz_sig, unsigned int *nz_hsig, unsigned int *nz_hval) {
	float y; // acumula soma das arestas
	__shared__ int nfol; // numero de folhas da arvore
	int j; // indice para thread ativa
	int a, b; // usados no calculo da faixa de elementos (da matriz triangular) a serem considerados
	unsigned int sig1, sig2, sig3, sig4; // assinaturas de tres nos - da o caminho em bits ate o raiz
	int bit; // contem bit sendo analizado
	int ancc; // indice do ancestral comum
	int nthreads; // numero de threads ativas
	int r, c; // linha e coluna da matriz triangular superior
	int bits; // conta quantos bits sao iguais
	int i = threadIdx.x; // identificador da thread
	int ib = blockIdx.x; // identificador do bloco
	int it; // indice de acesso global das threads
	int ennos; // tamanho da matriz de distancias
	
	nfol = nnos / 2; // folhas estao na metade inferior
	ennos = (nfol * (nfol - 1)) / 2;
	it = i + ib*nnos;
	
	if (i < nfol) { // nos folhos calculam distancia ate a raiz e armazena o caminho (assinatura
					// em bits) até a raiz
		y = 0;
		j = it;  // associa threads com nos folhas
		nz_sig[it] = 1;
		while (j != -1) {
			y = y + nz_br[j]; // acumula a distancia
			if (nz_p[j] == -1) 
				break;
			nz_sig[it] <<= 1;  // acumula o caminho
			if (nz_f1[nz_p[j]] == j) // acrescenta 0 se vier da direita (f2)
				nz_sig[it]++;         // ou 1 se vier da esquerda (f1)
			j = nz_p[j];
		}
		quadratic_probing_insert(nz_hsig, nz_hval, nz_sig[it], it, hnnos);
		nz_dr[it] = y;
	}

	__syncthreads(); // espera todas as threads chegarem até aqui

	if (i < (nfol-1)) { // nos internos calculam distancia ate a raiz e armazena o caminho
						// (assinatura em bits) até o raiz
		y = 0;
		j = it+nfol+1;  // associa threads com os nos internos
		nz_sig[j] = 1;
		if (nz_p[j] == -1) j = -1;
		while (j != -1) {
			y = y + nz_br[j]; // acumula a distancia
			if (nz_p[j] == -1) 
				break;
			nz_sig[it+nfol+1] <<= 1; // acumula o caminho
			if (nz_f1[nz_p[j]] == j)    // acrescenta 0 se vier da direita (f2)
				nz_sig[it+nfol+1]++; // ou 1 se vier da esquerda (f1)
			j = nz_p[j];
		}
		quadratic_probing_insert(nz_hsig, nz_hval, nz_sig[it+nfol+1], (it+nfol+1), hnnos);
		nz_dr[it+nfol+1] = y;
	}
	
	__syncthreads(); // espera todas as threads chegarem até aqui

	// se nfol (numero de especies) for impar, usamos nfol threads
	// se nfol (numero de especies) for par, usamos nfol-1 threads
	// isso evita termos que tratar de elementos restantes
	
	if ( (nfol % 2) == 0) {
		nthreads = nfol - 1; // nfol é par: cada thread calcula nfol/2 distancias 
		a = nfol / 2; // quantidade de elementos por thread
	} else {
		nthreads = nfol;	 // nfol é ímpar: cada thread calcula (nfol-1)/2 distancias
		a = (nfol - 1) / 2;  // quantidade de elementos por thread
	}

	if (i < nthreads) {
		for( b = i*a; b < a+(i*a); b++) {
			r = row_index(b, nfol);
			c = column_index(b, nfol);
			sig1 = nz_sig[r+ib*nnos];
			sig2 = nz_sig[c+ib*nnos];
			sig3 = 1; // inicia com 1 para diferenciar das demais assinaturas, i.e., 10, 100 etc
			bits = 0; // conta quantos bits sao iguais
			sig4 = 1; // recebe assinatura invertida
			while ( (sig1 & 1) == (sig2 & 1) && bits < 32) { // compara bit menos significativo
				bit = (sig1 & 1);
				bits++;
				sig1 >>= 1; // avanca para proximo bit
				sig2 >>= 1; // avanca para proximo bit
				sig3 <<= 1; // armazena bits coincidentes - caminho do ancestral comum
				if (bit) 
					sig3++; // soma 1 ou 0
			}
			while (bits>0) { // inverte a assinatura coincidente incluindo um 1 mais a esquerda
				sig4 <<= 1;
				if (sig3 & 1)
					sig4++;
				sig3 >>= 1;
				bits--;
			}
			ancc = quadratic_probing_search(nz_hsig, nz_hval, sig4, hnnos);
			nz_de[b+ib*ennos] = nz_dr[r+ib*nnos] + nz_dr[c+ib*nnos] - 2*nz_dr[ancc];
		}
	}
}





/*
Calcular o I de Moran para cada classe. Sao diversas arvores, cada uma tera o I de Moran para cada classe (nz_class),
em seguida faz-se a media e calcula a variancia entre elas.

Return: I de Moran por classe e a variancia para cada classe.
*/
__global__ void I_moran_Gpu(int nnos, int nrClass, float *nz_de, float *nz_trait, float *nz_class_range, float *nz_class_value, float MeanY, float Variance){
  	int d, r, c, a, b;
	int nfol, nthreads;
 	float SumProdCross, SumW, w;
  	short int p;
	int i = threadIdx.x; // identificador da thread
	int ib = blockIdx.x; // identificador do bloco
	int ennos;
	int base;
	__shared__ float sumTotal, sumTotalProdCross;
	extern __shared__ float nzClass[];

	for(d=0;d<nrClass;d++){
		nzClass[d] = nz_class_range[d];
	}

	nfol = nnos/2;
	ennos = (nfol * (nfol - 1)) / 2;
	base = ib * ennos;

	SumW = 0;

	if ( (nfol % 2) == 0) {
		nthreads = nfol - 1; // nfol é par: cada thread calcula nfol/2 distancias 
		a = nfol / 2; // quantidade de elementos por thread
	} else {
		nthreads = nfol;	 // nfol é ímpar: cada thread calcula (nfol-1)/2 distancias
		a = (nfol - 1) / 2;  // quantidade de elementos por thread
	}

	w = 1;
      	p = 2; //Symetric

	//Inicializa variaveis compartilhadas
	sumTotalProdCross = 0;
	sumTotal = 0;
	__syncthreads();//aguarda inicializacao das variaveis para continuar execução
	if (i < nthreads) {
  		for(d=0;d<nrClass;d++){
    			SumProdCross = 0;
	    		SumW = 0;
	
			for( b = i*a; b < a+(i*a); b++) {
				if (nz_de[b+base] > nzClass[d] && nz_de[b+base] <= nzClass[d+1]){
					r = row_index(b, nfol);
					c = column_index(b, nfol);
			
					SumW += (w*p);
					SumProdCross += (((nz_trait[r] - MeanY) * (nz_trait[c] - MeanY))*p);
				}
			}

			//Utilizar operacao atomica
				atomicFloatAdd(&sumTotalProdCross, SumProdCross);
				atomicFloatAdd(&sumTotal, SumW);
			__syncthreads(); // espera todas as threads chegarem até aqui
			//apenas uma thread calcula o I de Moran
			if (threadIdx.x == 0){ 
				nz_class_value[(ib*nrClass)+d] = (nfol / sumTotal) * (sumTotalProdCross / Variance);       // I de Moran
				sumTotalProdCross = 0;
				sumTotal = 0;
			}
			__syncthreads(); // espera todas as threads chegarem até aqui
   		}
	}
}


__device__ inline void atomicFloatAdd(float *address, float val)
{
	int tmp0 = *address;
	int i_val = __float_as_int(val + __int_as_float(tmp0));
	int tmp1;
	// compare and swap v = (old == tmp0) ? i_val : old;
	// returns old 
	while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0 )
	{
		tmp0 = tmp1;
		i_val = __float_as_int(val + __int_as_float(tmp1));
	}
}
